#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvm_ctrl.h>
#include <nvm_types.h>
#include <nvm_queue.h>
#include <nvm_util.h>
#include <nvm_admin.h>
#include <nvm_error.h>
#include <nvm_cmd.h>
#include <string>
#include <stdexcept>
#include <vector>
#include <cstdio>
#include <cstdint>
#include <cstring>
#include <map>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <ctrl.h>
#include <buffer.h>
#include <event.h>
#include <queue.h>
#include <nvm_parallel_queue.h>
#include <nvm_io.h>
#include <page_cache.h>
#include <util.h>
#include <iostream>
#include <fstream>
#include <byteswap.h>
#ifdef __DIS_CLUSTER__
#include <sisci_api.h>
#endif
#include "read_write_so.h"

#define READ 0
#define WRITE 1
#define MIXED 2
#define VERIFY 3

using error = std::runtime_error;
using std::string;

 //const char* const ctrls_paths[] = {"/dev/libnvm0", "/dev/libnvm1", "/dev/libnvm2", "/dev/libnvm3", "/dev/libnvm4", "/dev/libnvm5", "/dev/libnvm6"};
const char* const ctrls_paths[] = {"/dev/libnvm0"};

/*
__device__ void read_data(page_cache_t* pc, QueuePair* qp, const uint64_t starting_lba, const uint64_t n_blocks, const unsigned long long pc_entry) {
    //uint64_t starting_lba = starting_byte >> qp->block_size_log;
    //uint64_t rem_bytes = starting_byte & qp->block_size_minus_1;
    //uint64_t end_lba = CEIL((starting_byte+num_bytes), qp->block_size);

    //uint16_t n_blocks = CEIL(num_bytes, qp->block_size, qp->block_size_log);
 
    nvm_cmd_t cmd;
    uint16_t cid = get_cid(&(qp->sq));
    //printf("cid: %u\n", (unsigned int) cid);
 
    nvm_cmd_header(&cmd, cid, NVM_IO_READ, qp->nvmNamespace);
    uint64_t prp1 = pc->prp1[pc_entry];
    uint64_t prp2 = 0;
    if (pc->prps)
        prp2 = pc->prp2[pc_entry];
    //printf("tid: %llu\tstart_lba: %llu\tn_blocks: %llu\tprp1: %p\n", (unsigned long long) threadIdx.x, (unsigned long long) starting_lba, (unsigned long long) n_blocks, (void*) prp1);
    nvm_cmd_data_ptr(&cmd, prp1, prp2);
    nvm_cmd_rw_blks(&cmd, starting_lba, n_blocks);
    uint16_t sq_pos = sq_enqueue(&qp->sq, &cmd);

    uint32_t cq_pos = cq_poll(&qp->cq, cid);
    sq_dequeue(&qp->sq, sq_pos);
    cq_dequeue(&qp->cq, cq_pos);
 
    put_cid(&qp->sq, cid);
 
}
*/

__global__
void sequential_access_kernel(Controller** ctrls, page_cache_d_t* pc,  uint32_t req_size, uint32_t n_reqs, //unsigned long long* req_count,
                                uint32_t num_ctrls, uint64_t reqs_per_thread, uint32_t access_type, uint64_t s_offset, uint64_t o_offset){
    //printf("in threads\n");
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    // uint32_t bid = blockIdx.x;
    // uint32_t smid = get_smid();

    uint32_t ctrl = (tid/32) % (num_ctrls);
    uint32_t queue = (tid/32) % (ctrls[ctrl]->n_qps);
    uint64_t itr=0; 

//    printf("Num pages: %llu, s_offset: %llu n_reqs: %llu\t req_size: %llu\n", (unsigned long long int) pc->n_pages, (unsigned long long int) s_offset, (unsigned long long int) n_reqs, (unsigned long long) req_size); 
    for (;tid < pc->n_pages; tid = tid+n_reqs){
            uint64_t start_block = (o_offset+s_offset + tid*req_size) >> ctrls[ctrl]->d_qps[queue].block_size_log ;
            uint64_t pc_idx = (tid);
            //uint64_t start_block = (tid*req_size) >> ctrls[ctrl]->d_qps[queue].block_size_log;
            //start_block = tid;
            uint64_t n_blocks = req_size >> ctrls[ctrl]->d_qps[queue].block_size_log; /// ctrls[ctrl].ns.lba_data_size;;
//            printf("itr:%llu\ttid: %llu\tstart_block: %llu\tn_blocks: %llu\tpc_idx: %llu\n", (unsigned long long)itr, (unsigned long long) tid, (unsigned long long) start_block, (unsigned long long) n_blocks, (unsigned long long) pc_idx);
            itr = itr+1; 
            // uint8_t opcode;
            // for (size_t i = 0; i < reqs_per_thread; i++) {
                if (access_type == READ) {
                    read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, pc_idx);
                    //if(tid ==( pc->n_pages - 1)){
                    //        printf("I am here\n");
                    //        hexdump(pc->base_addr+tid*req_size, 4096); 
                    //}
                }
                else {
                    write_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, pc_idx);
                }

            
            // }
            //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
            //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
            //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
            //__syncthreads();
            //read_data(pc, (ctrls[ctrl].d_qps)+(queue),start_block*2, n_blocks, tid);
            //printf("tid: %llu finished\n", (unsigned long long) tid);
    }
}

/*__global__
void random_access_kernel(Controller** ctrls, page_cache_t* pc,  uint32_t req_size, uint32_t n_reqs, unsigned long long* req_count, uint32_t num_ctrls, uint64_t* assignment, uint64_t reqs_per_thread, uint32_t access_type, uint8_t* access_type_assignment) {
    //printf("in threads\n");
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bid = blockIdx.x;
    uint32_t smid = get_smid();

    uint32_t ctrl = (tid/32) % (num_ctrls);
    uint32_t queue = (tid/32) % (ctrls[ctrl]->n_qps);


    if (tid < n_reqs) {
        uint64_t start_block = (assignment[tid]*req_size) >> ctrls[ctrl]->d_qps[queue].block_size_log;
        //uint64_t start_block = (tid*req_size) >> ctrls[ctrl]->d_qps[queue].block_size_log;
        //start_block = tid;
        uint64_t n_blocks = req_size >> ctrls[ctrl]->d_qps[queue].block_size_log; /// ctrls[ctrl].ns.lba_data_size;;
        //printf("tid: %llu\tstart_block: %llu\tn_blocks: %llu\n", (unsigned long long) tid, (unsigned long long) start_block, (unsigned long long) n_blocks);

        uint8_t opcode;
        for (size_t i = 0; i < reqs_per_thread; i++) {
            if (access_type == MIXED) {
                opcode = access_type_assignment[tid];
                access_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid, opcode);
            }
            else if (access_type == READ) {
                read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);

            }
            else {
                write_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
            }
        }
        //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
        //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
        //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
        //__syncthreads();
        //read_data(pc, (ctrls[ctrl].d_qps)+(queue),start_block*2, n_blocks, tid);
        //printf("tid: %llu finished\n", (unsigned long long) tid);

    }

}
*/

__global__ 
void verify_kernel(uint64_t* orig_h, uint64_t* nvme_h, uint64_t n_elems,uint32_t n_reqs){
        uint64_t tid = blockIdx.x*blockDim.x + threadIdx.x; 

        for (;tid < n_elems; tid = tid+n_reqs){
           uint64_t orig_val = orig_h[tid]; 
           uint64_t nvme_val = nvme_h[tid]; 
           if(orig_val != nvme_val)
              printf("MISMATCH: at %llu\torig_val:%llu\tnvme_val:%llu\tn_reqs:%lu\tn_elms:%llu\n",tid, (unsigned long long)orig_val, (unsigned long long)nvme_h, n_reqs, n_elems);
        }
        __syncthreads();//really not needed. 
}


uint32_t nvme_dev_setting(uint32_t cudaDevice)
{
    hipDeviceProp_t properties;
    if (hipGetDeviceProperties(&properties, cudaDevice) != hipSuccess)
    {
        fprintf(stderr, "Failed to get CUDA device properties\n");
        return 1;
    }

    cuda_err_chk(hipSetDevice(cudaDevice));
    std::vector<Controller*> ctrls(1);
    for (size_t i = 0 ; i < 1; i++)
        ctrls[i] = new Controller(ctrls_paths[i], 1, cudaDevice, 1024, 1);
}

  uint32_t nvme_dev_read()
  {
    fprintf(stdout, "read\n");

    return 1;
  }

  uint32_t nvme_dev_write()
  {
    fprintf(stdout, "write\n");

    return 2;
  }