#include "hip/hip_runtime.h"
#include "read_write_so.h"

#define READ 0
#define WRITE 1
#define MIXED 2
#define VERIFY 3

using error = std::runtime_error;
using std::string;

 //const char* const ctrls_paths[] = {"/dev/libnvm0", "/dev/libnvm1", "/dev/libnvm2", "/dev/libnvm3", "/dev/libnvm4", "/dev/libnvm5", "/dev/libnvm6"};

/*
__device__ void read_data(page_cache_t* pc, QueuePair* qp, const uint64_t starting_lba, const uint64_t n_blocks, const unsigned long long pc_entry) {
    //uint64_t starting_lba = starting_byte >> qp->block_size_log;
    //uint64_t rem_bytes = starting_byte & qp->block_size_minus_1;
    //uint64_t end_lba = CEIL((starting_byte+num_bytes), qp->block_size);

    //uint16_t n_blocks = CEIL(num_bytes, qp->block_size, qp->block_size_log);
 
    nvm_cmd_t cmd;
    uint16_t cid = get_cid(&(qp->sq));
    //printf("cid: %u\n", (unsigned int) cid);
 
    nvm_cmd_header(&cmd, cid, NVM_IO_READ, qp->nvmNamespace);
    uint64_t prp1 = pc->prp1[pc_entry];
    uint64_t prp2 = 0;
    if (pc->prps)
        prp2 = pc->prp2[pc_entry];
    //printf("tid: %llu\tstart_lba: %llu\tn_blocks: %llu\tprp1: %p\n", (unsigned long long) threadIdx.x, (unsigned long long) starting_lba, (unsigned long long) n_blocks, (void*) prp1);
    nvm_cmd_data_ptr(&cmd, prp1, prp2);
    nvm_cmd_rw_blks(&cmd, starting_lba, n_blocks);
    uint16_t sq_pos = sq_enqueue(&qp->sq, &cmd);

    uint32_t cq_pos = cq_poll(&qp->cq, cid);
    sq_dequeue(&qp->sq, sq_pos);
    cq_dequeue(&qp->cq, cq_pos);
 
    put_cid(&qp->sq, cid);
 
}
*/

__global__
void sequential_access_kernel(Controller** ctrls, page_cache_d_t* pc,  uint32_t req_size, uint32_t n_reqs, //unsigned long long* req_count,
                                uint32_t num_ctrls, uint64_t reqs_per_thread, uint32_t access_type, uint64_t s_offset, uint64_t o_offset){
    //printf("in threads\n");
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    // uint32_t bid = blockIdx.x;
    // uint32_t smid = get_smid();

    uint32_t ctrl = (tid/32) % (num_ctrls);
    uint32_t queue = (tid/32) % (ctrls[ctrl]->n_qps);
    uint64_t itr=0; 

//    printf("Num pages: %llu, s_offset: %llu n_reqs: %llu\t req_size: %llu\n", (unsigned long long int) pc->n_pages, (unsigned long long int) s_offset, (unsigned long long int) n_reqs, (unsigned long long) req_size); 
    for (;tid < pc->n_pages; tid = tid+n_reqs){
            uint64_t start_block = (o_offset+s_offset + tid*req_size) >> ctrls[ctrl]->d_qps[queue].block_size_log ;
            uint64_t pc_idx = (tid);
            //uint64_t start_block = (tid*req_size) >> ctrls[ctrl]->d_qps[queue].block_size_log;
            //start_block = tid;
            uint64_t n_blocks = req_size >> ctrls[ctrl]->d_qps[queue].block_size_log; /// ctrls[ctrl].ns.lba_data_size;;
//            printf("itr:%llu\ttid: %llu\tstart_block: %llu\tn_blocks: %llu\tpc_idx: %llu\n", (unsigned long long)itr, (unsigned long long) tid, (unsigned long long) start_block, (unsigned long long) n_blocks, (unsigned long long) pc_idx);
            itr = itr+1; 
            // uint8_t opcode;
            // for (size_t i = 0; i < reqs_per_thread; i++) {
                if (access_type == READ) {
                    read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, pc_idx);
                    //if(tid ==( pc->n_pages - 1)){
                    //        printf("I am here\n");
                    //        hexdump(pc->base_addr+tid*req_size, 4096); 
                    //}
                }
                else {
                    write_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, pc_idx);
                }

            
            // }
            //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
            //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
            //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
            //__syncthreads();
            //read_data(pc, (ctrls[ctrl].d_qps)+(queue),start_block*2, n_blocks, tid);
            //printf("tid: %llu finished\n", (unsigned long long) tid);
    }
}

/*__global__
void random_access_kernel(Controller** ctrls, page_cache_t* pc,  uint32_t req_size, uint32_t n_reqs, unsigned long long* req_count, uint32_t num_ctrls, uint64_t* assignment, uint64_t reqs_per_thread, uint32_t access_type, uint8_t* access_type_assignment) {
    //printf("in threads\n");
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bid = blockIdx.x;
    uint32_t smid = get_smid();

    uint32_t ctrl = (tid/32) % (num_ctrls);
    uint32_t queue = (tid/32) % (ctrls[ctrl]->n_qps);


    if (tid < n_reqs) {
        uint64_t start_block = (assignment[tid]*req_size) >> ctrls[ctrl]->d_qps[queue].block_size_log;
        //uint64_t start_block = (tid*req_size) >> ctrls[ctrl]->d_qps[queue].block_size_log;
        //start_block = tid;
        uint64_t n_blocks = req_size >> ctrls[ctrl]->d_qps[queue].block_size_log; /// ctrls[ctrl].ns.lba_data_size;;
        //printf("tid: %llu\tstart_block: %llu\tn_blocks: %llu\n", (unsigned long long) tid, (unsigned long long) start_block, (unsigned long long) n_blocks);

        uint8_t opcode;
        for (size_t i = 0; i < reqs_per_thread; i++) {
            if (access_type == MIXED) {
                opcode = access_type_assignment[tid];
                access_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid, opcode);
            }
            else if (access_type == READ) {
                read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);

            }
            else {
                write_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
            }
        }
        //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
        //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
        //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
        //__syncthreads();
        //read_data(pc, (ctrls[ctrl].d_qps)+(queue),start_block*2, n_blocks, tid);
        //printf("tid: %llu finished\n", (unsigned long long) tid);

    }

}
*/

__global__
void random_access_kernel(Controller** ctrls, page_cache_d_t* pc,  uint32_t req_size, uint32_t n_reqs, uint32_t num_ctrls, uint64_t* assignment, uint64_t reqs_per_thread, uint32_t access_type) {
    //printf("in threads\n");
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bid = blockIdx.x;
    uint32_t smid = get_smid();

    uint32_t ctrl = (tid/32) % (num_ctrls);
    uint32_t queue = (tid/32) % (ctrls[ctrl]->n_qps);


    if (tid < n_reqs) {
        uint64_t start_block = (assignment[tid]*req_size) >> ctrls[ctrl]->d_qps[queue].block_size_log;
        //uint64_t start_block = (tid*req_size) >> ctrls[ctrl]->d_qps[queue].block_size_log;
        //start_block = tid;
        uint64_t n_blocks = req_size >> ctrls[ctrl]->d_qps[queue].block_size_log; /// ctrls[ctrl].ns.lba_data_size;;
        //printf("tid: %llu\tstart_block: %llu\tn_blocks: %llu\n", (unsigned long long) tid, (unsigned long long) start_block, (unsigned long long) n_blocks);

        for (size_t i = 0; i < reqs_per_thread; i++) {
            if (access_type == READ) {
                read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);

            }
            else {
                write_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
            }
        }
        //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
        //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
        //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
        //__syncthreads();
        //read_data(pc, (ctrls[ctrl].d_qps)+(queue),start_block*2, n_blocks, tid);
        //printf("tid: %llu finished\n", (unsigned long long) tid);

    }

}

__global__ 
void verify_kernel(uint64_t* orig_h, uint64_t* nvme_h, uint64_t n_elems,uint32_t n_reqs){
        uint64_t tid = blockIdx.x*blockDim.x + threadIdx.x; 

        for (;tid < n_elems; tid = tid+n_reqs){
           uint64_t orig_val = orig_h[tid]; 
           uint64_t nvme_val = nvme_h[tid]; 
           if(orig_val != nvme_val)
              printf("MISMATCH: at %llu\torig_val:%llu\tnvme_val:%llu\tn_reqs:%lu\tn_elms:%llu\n",tid, (unsigned long long)orig_val, (unsigned long long)nvme_h, n_reqs, n_elems);
        }
        __syncthreads();//really not needed. 
}


int dev_set(uint32_t cudaDevice, void* src_in)
{

    int fd_in;
    char* input_f = (char*) src_in;

    if((fd_in = open(input_f, O_RDONLY)) == -1)
    {
        fprintf(stderr, "Input file cannot be opened!\n");
        return 1;
    }
    fstat(fd_in, &sb_in);

    printf("sb_in.st_size: %ld\n", sb_in.st_size);

    map_in = mmap(NULL, sb_in.st_size, PROT_READ, MAP_SHARED, fd_in, 0);
    if(map_in == (void*)-1)
    {
        fprintf(stderr, "Input file map failed %d\n", map_in);
        return 1;
    }

    n_tsteps = ceil((float)(sb_in.st_size-0)/(float)total_cache_size);
    n_telem = ((sb_in.st_size-0)/sizeof(int64_t));

    // for(int id=0; id<30; id++)
    // {
    //     printf("id[%d] = %.3f\n", id, ((float*)map_in)[id]);
    // }

    hipDeviceProp_t properties;
    if (hipGetDeviceProperties(&properties, cudaDevice) != hipSuccess)
    {
        fprintf(stderr, "Failed to get CUDA device properties\n");
        return 1;
    }

    cuda_err_chk(hipSetDevice(cudaDevice));
    for (size_t i = 0 ; i < 1; i++)
        ctrls[i] = new Controller(ctrls_paths[i], 1, cudaDevice, queueDepth, queueNum);
    fprintf(stdout, "controller created\n");
    
    char st[15];
    cuda_err_chk(hipDeviceGetPCIBusId(st, 15, cudaDevice));
    fprintf(stdout, "cudaDevice pcie: %s\n", st);

    h_pc = new page_cache_t(page_size, n_pages, cudaDevice, ctrls[0][0], (uint64_t) 64, ctrls);
    fprintf(stdout, "finished creating cache\n Total Cache size (MBs): %.2f\n", ((float)total_cache_size/(1024*1024)));

     //QueuePair* d_qp;
    d_pc = (page_cache_d_t*) (h_pc->d_pc_ptr);
    printf("n_tsteps: %lu, n_telem: %llu, n_pages:%llu\n", n_tsteps, n_telem, n_pages);

    return 0;
}

int nvme_dev_write()
{
    // strat write
    for (uint32_t cstep =0; cstep < n_tsteps; cstep++) {
        uint64_t cpysize = std::min(total_cache_size, sb_in.st_size-s_offset);
        printf("cstep: %lu  s_offset: %llu   cpysize: %llu pcaddr:%p, block size: %llu, grid size: %llu\n", cstep, s_offset, cpysize, h_pc->pdt.base_addr, b_size, g_size);

        cuda_err_chk(hipMemcpy(h_pc->pdt.base_addr, map_in+s_offset+0, cpysize, hipMemcpyHostToDevice));

        hipEventCreate(&start_write); 
        hipEventCreate(&stop_write);
        hipEventRecord(start_write, 0);
        sequential_access_kernel<<<g_size, b_size>>>(h_pc->pdt.d_ctrls, d_pc, page_size, n_threads, //d_req_count,
        1, 1, WRITE, s_offset, 0);

        cuda_err_chk(hipDeviceSynchronize());
        hipEventRecord(stop_write, 0);
        hipEventSynchronize(stop_write);

        float wcompleted = 100*(total_cache_size*(cstep+1))/(sb_in.st_size);
        hipEventElapsedTime(&welapsed, start_write, stop_write);
        std::cout << "Write Completed:" << wcompleted << "%   Write Time:" <<welapsed << "ms" << std::endl;

         s_offset = s_offset + cpysize;
    }

    return 0;
}

uint64_t nvme_dev_read(void* read_offset, uint64_t idNUM, uint64_t read_size)
{
    // start read
    uint64_t* local_read_offset = (uint64_t*)read_offset;
    // n_tsteps = ceil((float)(read_size)/(float)total_cache_size);
    // for (uint32_t cstep =0; cstep < n_tsteps; cstep++) {
    // for (uint32_t cstep = 0; cstep < idNUM; cstep ++) {
        // uint64_t cpysize = std::min(total_cache_size, read_size);

        uint64_t* d_assignment;
        threadNum = idNUM;
        g_size = (threadNum + b_size - 1)/b_size;
        n_threads = b_size * g_size;
        cuda_err_chk(hipMalloc(&d_assignment, n_threads*sizeof(uint64_t)));
        cuda_err_chk(hipMemcpy(d_assignment, local_read_offset,  n_threads*sizeof(uint64_t), hipMemcpyHostToDevice));

        // uint64_t data_read_offset = local_read_offset[cstep] * read_size;
        // printf("cstep: %lu  s_offset: %llu   cpysize: %llu pcaddr:%p, block size: %llu, grid size: %llu\n", cstep, s_offset, cpysize, h_pc->pdt.base_addr, b_size, g_size);

        cuda_err_chk(hipMemset(h_pc->pdt.base_addr, 0, total_cache_size));
        
        hipEventCreate(&start_read); 
        hipEventCreate(&stop_read);
        hipEventRecord(start_read, 0);
        // sequential_access_kernel<<<g_size, b_size>>>(h_pc->pdt.d_ctrls, d_pc, page_size, n_threads, //d_req_count,
        // 1, 1, READ, data_read_offset, 0);

        random_access_kernel<<<g_size, b_size>>>(h_pc->pdt.d_ctrls, d_pc, page_size,
        n_threads, 1, d_assignment, 1, READ);
        
        cuda_err_chk(hipDeviceSynchronize());

        hipEventRecord(stop_read, 0);
        hipEventSynchronize(stop_read);
        
        // float rcompleted = 100*(cpysize*(cstep+1))/(read_size);
        hipEventElapsedTime(&relapsed, start_read, stop_read);
        // std::cout << "Read Completed:" << rcompleted << "%   Read Time:" <<relapsed << "ms" << std::endl;
        std::cout << "Read Time:" <<relapsed << "ms" << std::endl;

        // local_read_offset = local_read_offset + cpysize;

        // printf("cuda addr: %p\n", h_pc->pdt.base_addr);
    // }

    return reinterpret_cast<uint64_t>(h_pc->pdt.base_addr);
}

int free_dev()
{
    for (size_t i = 0; i < 1; i++)
            delete ctrls[i];
    delete h_pc;

    return 0;
}
